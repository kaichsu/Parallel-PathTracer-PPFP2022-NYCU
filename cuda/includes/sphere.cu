#include "hip/hip_runtime.h"
#ifndef SPHERE_H
#define SPHERE_H

#include "hittable.cu"
#include "vec3.cu"

class sphere : public hittable {
    public:
        __device__ sphere() {}
        __device__ sphere(point3 cen, float r, material* m)
            : center(cen), radius(r), mat_ptr(m) {};

        __device__ virtual bool hit(
            const ray& r, const float &t_min, const float &t_max, hit_record& rec) const override;

    public:
        point3 center;
        float radius;
        material *mat_ptr;
};

__device__ bool sphere::hit(const ray& r, const float &t_min, const float &t_max, hit_record& rec) const {
    vec3 oc = r.origin() - center;
    auto a = r.direction().length_squared();
    auto half_b = dot(oc, r.direction());
    auto c = oc.length_squared() - radius*radius;

    auto discriminant = half_b*half_b - a*c;
    if (discriminant < 0) return false;
    auto sqrtd = sqrtf(discriminant);

    // Find the nearest root that lies in the acceptable range.
    auto root = (-half_b - sqrtd) / a;
    if (root < t_min || t_max < root) {
        root = (-half_b + sqrtd) / a;
        if (root < t_min || t_max < root)
            return false;
    }

    rec.t = root;
    rec.p = r.at(rec.t);
    
    rec.normal = (rec.p - center) / radius;

    vec3 outward_normal = (rec.p - center) / radius;
    rec.set_face_normal(r, outward_normal);
    rec.mat_ptr = mat_ptr;

    return true;
}

#endif